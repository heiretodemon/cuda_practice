#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd(int n, float *a, float *b, float *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] + b[i];
    }
}

int main(){
    int N = 1024;
    float *a, *b, *c;
    float *devA, *devB, *devC;
    a = (float*)malloc(N * sizeof(float));
    b = (float*)malloc(N * sizeof(float));
    c = (float*)malloc(N * sizeof(float));
    hipMalloc(&devA, N*sizeof(float));
    hipMalloc(&devB, N*sizeof(float));
    hipMalloc(&devC, N*sizeof(float));

    memset(c, 0, N*sizeof(float));
    // memset(a, 1, N*sizeof(float));
    // memset(b, 2, N*sizeof(float));
    hipMemcpy(devA, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devB, b, N*sizeof(float), hipMemcpyHostToDevice);

    vecAdd <<< (N+255)/256, 256 >>> (N, devA, devB, devC);
    hipMemcpy(c, devC, N*sizeof(float), hipMemcpyDeviceToHost);
    // for (int i=0;i<10;i++){
    //     printf("%f, %f", a[0], a[1]);
    //     printf("%f\n", c[i]);
    // }
    return 0;
}
